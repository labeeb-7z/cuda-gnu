#include "hip/hip_runtime.h"
#include "gpu.h"
__global__ void multiply(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] * b[i];
}

void gpumul(int *a, int *b, int *c, int n)
{
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, n * sizeof(int));
    hipMalloc((void **)&dev_b, n * sizeof(int));
    hipMalloc((void **)&dev_c, n * sizeof(int));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(10, 1);
    dim3 dimGrid(1, 1);
    multiply<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
