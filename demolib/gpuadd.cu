#include "hip/hip_runtime.h"
#include "gpu.h"
#include <iostream>
using namespace std;
__global__ void add(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
    //cout<<index<<endl;
}

void gpuadd(int *a, int *b, int *c, int n)
{
    //cout<<"gpuadd"<<endl;
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, n * sizeof(int));
    hipMalloc((void **)&dev_b, n * sizeof(int));
    hipMalloc((void **)&dev_c, n * sizeof(int));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(10, 1,1);
    dim3 dimGrid(1, 1);

    add<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}