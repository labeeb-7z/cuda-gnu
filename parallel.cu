#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <gnuastro/fits.h>

__global__ void gpu_kernel(float *device_array, int image_height, int image_width)
{

    //get the current block and thread id
    int bID = blockIdx.x + blockIdx.y * gridDim.x;
    int tID = bID * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    //define the stride -> total number of threads
    int stride = blockDim.x * blockDim.y * gridDim.x * gridDim.y;


    //iterate over all pixels which are to be processed by this thread
    for (int i = tID; i < image_height * image_width; i += stride)
    {

        // get the row and column of the current pixel
        int row = tID / image_width;
        int col = tID % image_width;

        printf("pixel (%d, %d) on thread (%d, %d) on block (%d, %d) : %f\n", 
                            row, col, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, device_array[i]);
    }
}

int main(void)
{
    // Read the image
    gal_data_t *image;
    char *filename = "horsehead.fits", *hdu = "0";

    image = gal_fits_img_read_to_type(filename, hdu, GAL_TYPE_FLOAT32,
                                      -1, 1);

    int image_height = image->dsize[0];
    int image_width = image->dsize[1];


    // Allocate memory on the device(GPU)
    float *device_array;
    hipMalloc((void **)&device_array, image_height * image_width * sizeof(float));
    hipMemcpy(device_array,(float *) image->array, image_height * image_width * sizeof(float), hipMemcpyHostToDevice);


    // define the number of threads in a block and the number of blocks in a grid
    dim3 block(32, 32);
    int block_height = image_height / block.y +1;
    int block_width = image_width / block.x +1;
    dim3 grid(block_width, block_height);


    // launch the kernel
    gpu_kernel<<<grid, block>>>(device_array, image_height, image_width);
   
    
    hipFree(device_array);
    gal_data_free(image);
    return EXIT_SUCCESS;
}