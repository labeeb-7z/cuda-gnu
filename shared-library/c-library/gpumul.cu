#include "hip/hip_runtime.h"
#include "gpu.h"
__global__ void multiply_kernel(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] * b[i];
}

__global__ void mulby2_kernel(int *a, int *b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        b[i] = a[i] * 2;
}

void gpumul(int *a, int *b, int *c, int n)
{
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, n * sizeof(int));
    hipMalloc((void **)&dev_b, n * sizeof(int));
    hipMalloc((void **)&dev_c, n * sizeof(int));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(1000, 1);
    dim3 dimGrid(1, 1);
    multiply_kernel<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
